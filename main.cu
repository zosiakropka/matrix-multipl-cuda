
#include <hip/hip_runtime.h>
/* 
 * File:   main.cu
 * Author: Zosia Sobocinska
 *
 * Created on November 20, 2013, 7:17 PM
 */


#define BLOCK_SIZE 4

__global__ void multiple(float* C, float* A, float* B, uint size) {

    int x = threadIdx.x;
    int y = threadIdx.y;

    float C_yx = 0;
    for (uint i = 0; i < size; ++i) {
        float A_yi = A[y * size + i];
        float B_ix = B[i * size + x];
        C_yx += A_yi * B_ix;
    }

    C[y * size + x] = C_yx;
}

int main(int argc, char** argv) {

    //////////////////////////////////////////////////////////////////////////
    // USAGE

    if (argc != 3) {
        printf("Usage: %s <dummy> <matrix_size N=width=height>", argv[0]);
    }
    char time_msg[20];
    const char* testrun = getenv("TESTRUN");
    if (testrun != NULL && (*testrun) == 'y') {
        strcpy(time_msg, "%d");
    } else {
        strcpy(time_msg, "%dms\n");
    }

    uint size = atoi(argv[2]);

}
