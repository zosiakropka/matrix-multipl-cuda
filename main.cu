#include "hip/hip_runtime.h"
/* 
 * File:   main.cu
 * Author: Zosia Sobocinska
 *
 * Created on November 20, 2013, 7:17 PM
 */

#include <stdio.h>
#include "mt19937-64.c"

#define BLOCK_SIZE 4

__global__ void multiple(float* C, float* A, float* B, uint size) {

    int x = threadIdx.x;
    int y = threadIdx.y;

    float C_yx = 0;
    for (uint i = 0; i < size; ++i) {
        float A_yi = A[y * size + i];
        float B_ix = B[i * size + x];
        C_yx += A_yi * B_ix;
    }

    C[y * size + x] = C_yx;
}

namespace mtrx {
    namespace host {

        void alloc_mem(float** matrix, uint size) {
            (*matrix) = (float*) malloc(size * size * sizeof (float));
        }

        void free_mem(float** matrix) {
            free((*matrix));
            (*matrix) = NULL;
        }

        void fill(float** matrix, uint size) {
            for (uint i = 0; i < size * size; ++i)
                (*matrix)[i] = (float) genrand64_real1();
        }

        void cuda_host2dev(float *host_matrix, float *dev_matrix, uint size) {
            hipMemcpy(dev_matrix, host_matrix, size*size, hipMemcpyHostToDevice);
        }

    }
    namespace dev {

        void alloc_mem(float** matrix, uint size) {
            hipMalloc((void**) matrix, size * size * sizeof (float));
        }

        void free_mem(float** matrix) {
            hipFree((*matrix));
            (*matrix) = NULL;
        }

        void cuda_dev2host(float* dev_matrix, float* host_matrix, uint size) {
            hipMemcpy(host_matrix, dev_matrix, size*size, hipMemcpyDeviceToHost);
        }
    }
}

/**
 */
int main(int argc, char** argv) {

    //////////////////////////////////////////////////////////////////////////
    // USAGE

    if (argc != 3) {
        printf("Usage: %s <dummy> <matrix_size N=width=height>", argv[0]);
    }
    char time_msg[20];
    const char* testrun = getenv("TESTRUN");
    if (testrun != NULL && (*testrun) == 'y') {
        strcpy(time_msg, "%d");
    } else {
        strcpy(time_msg, "%dms\n");
    }

    uint size = atoi(argv[2]);

    //////////////////////////////////////////////////////////////////////////
    // INPUT
    float* A_host;
    mtrx::host::alloc_mem(&A_host, size);
    mtrx::host::fill(&A_host, size);

    float* B_host;
    mtrx::host::alloc_mem(&B_host, size);
    mtrx::host::fill(&B_host, size);

    float* A_dev;
    mtrx::dev::alloc_mem(&A_dev, size);

    float* B_dev;
    mtrx::dev::alloc_mem(&B_dev, size);

    //////////////////////////////////////////////////////////////////////////
    // OUTPUT

    float* C_host;
    mtrx::host::alloc_mem(&C_host, size);

    float* C_dev;
    mtrx::dev::alloc_mem(&C_dev, size);



    //////////////////////////////////////////////////////////////////////////
    // SEND TO GPU

    mtrx::host::cuda_host2dev(A_host, A_dev, size);
    mtrx::host::cuda_host2dev(B_host, B_dev, size);

    // dimensions of the blocks
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    // dimensions of the grid
    dim3 grid(size / threads.x, size / threads.y);

    //////////////////////////////////////////////////////////////////////////
    // RETRIEVE FROM GPU

    mtrx::dev::cuda_dev2host(C_dev, C_host, size);

    //////////////////////////////////////////////////////////////////////////
    // FREE MEMORY

    mtrx::host::free_mem(&A_host);
    mtrx::host::free_mem(&B_host);
    mtrx::host::free_mem(&C_host);

    mtrx::dev::free_mem(&A_dev);
    mtrx::dev::free_mem(&B_dev);
    mtrx::dev::free_mem(&C_dev);

}
